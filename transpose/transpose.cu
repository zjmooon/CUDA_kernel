#include "hip/hip_runtime.h"
#include "../common.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCK_SIZE 32

// host端做矩阵转置，与device端的结果进行比较
void transposeHost(const int *src, int *dst, const int nrows, const int ncols)
{
    for (int iy = 0; iy < nrows; ++iy)
    {
        for (int ix = 0; ix < ncols; ++ix)
        {
            dst[(ix * nrows) + iy] = src[((iy * ncols) + ix)];
        }
    }
}

void verifyResult(const int *hostMatrix, const int *deviceMatrix, const int nrows, const int ncols)
{
    for (int iy = 0; iy < nrows; ++iy)
    {
        for (int ix = 0; ix < ncols; ++ix)
        {
            if (hostMatrix[(iy * ncols) + ix] != deviceMatrix[(iy * ncols) + ix]) {
                std::cout << "result does not match in " << iy << "," << ix << std::endl;
                return;
            }

        }
    }
    std::cout << "succees! result match" << std::endl;
}

void printMatrix(const int *matrix, const int nrows, const int ncols)
{
    for (int iy = 0; iy < nrows; ++iy)
    {
        for (int ix = 0; ix < ncols; ++ix)
        {
            std::cout << matrix[(iy * ncols) + ix] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}



// device navie
__global__ void kTransposeNavie(const int *src, int *dst, const int M, const int N)
{
    const int gx = blockDim.x * blockIdx.x + threadIdx.x;
    const int gy = blockDim.y * blockIdx.y + threadIdx.y;

    if (gx < N && gy < M) {
        dst[gx * M + gy] = src[gy * N + gx];
    }
    /* 
    * load合并访存，store未合并访存
    * sector:物理层面概念，最小的load/store单位，32 bytes
    * load/store(SIMT): (1 << 13) * (1 << 12) (thread) / 32(thread / warp) = 1048576(instrument) --> 1048576(request)
    * load单个请求: 32 (thread) * 4 (byte / thead) = 128 byte --> sectors/Req = 128 / 32 = 4
    */
}
void iTransposeNavie(const int *src, int *dst, const int M, const int N, int* kernel_result) {
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(CEIL(N, BLOCK_SIZE), CEIL(M, BLOCK_SIZE));

    kTransposeNavie<<<gridSize, blockSize>>>(src, dst, M, N);

    // hipMemcpy(kernel_result, dst, M * N * sizeof(int), hipMemcpyDeviceToHost);
    // printMatrix(kernel_result, N, M);
}



// 合并写入(因为有缓存读load的机制，可以尽量提升性能，写store没有缓存机制，读load可以在空间和时间层面上进行缓存实现)
__global__ void kTransposeStoreCoalesce(const int *src, int *dst, const int M, const int N)
{
    const int gx = blockDim.x * blockIdx.x + threadIdx.x;
    const int gy = blockDim.y * blockIdx.y + threadIdx.y;

    if (gx < M && gy < N) {
        dst[gy * M + gx] = __ldg(&src[gx * N + gy]);
    }
}
void iTransposeStoreCoalesce(const int *src, int *dst, const int M, const int N, int* kernel_result) {
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(CEIL(N, BLOCK_SIZE), CEIL(M, BLOCK_SIZE));

    kTransposeStoreCoalesce<<<gridSize, blockSize>>>(src, dst, M, N);
}


// 使用shared memory 实现内存在读写上的合并
template<const int blockSize>
__global__ void kTransposeSharedCoalesce(const int *src, int *dst, const int M, const int N)
{
    const int bx = blockIdx.x * blockSize;
    const int by = blockIdx.y * blockSize;
    const int gx = bx + threadIdx.x;
    const int gy = by + threadIdx.y;
    __shared__ int s_data[blockSize][blockSize]; 

    if (gx < N && gy < M) {
        s_data[threadIdx.y][threadIdx.x] = src[gy * N + gx]; // 全局内存合并读取
    }
    __syncthreads();

    const int gx2 = by + threadIdx.x;
    const int gy2 = bx + threadIdx.y;

    if (gx2 < M && gy2 < N) {
        dst[gy2 * M + gx2] = s_data[threadIdx.x][threadIdx.y]; 
        // 全局内存合并写入 但共享内存Bank Conflict
    }

/* 
* 定义：相同线程束中(更严谨:相同memory transaction(load/store 128 bytes)，即可能小于32 线程)的不同线程访问相同Bank的不同地址。(只针对于共享内存)
* 所以向量化对于Bank Conflict有影响，一个线程不同的数据load/store会有不同的Bank Conflict效果
* 一共有32 bank，每个Bank32 bits 数据，或者64 bits
* 避免Bank Conflict: 1.padding(改变共享内存在Bank的布局) 2.swizzling(异或特性) 3.如果可以，调整线程访问模式为广播模式 
* https://forums.developer.nvidia.com/t/how-to-understand-the-bank-conflict-of-shared-mem/260900/2
*/
}
void iTransposeSharedCoalesce(const int *src, int *dst, const int M, const int N, int* kernel_result) {
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(CEIL(N, BLOCK_SIZE), CEIL(M, BLOCK_SIZE));

    kTransposeSharedCoalesce<BLOCK_SIZE><<<gridSize, blockSize>>>(src, dst, M, N);
}


// 使用shared memory 实现内存在读写上的合并 + padding避免Bank Conclict
template<const int blockSize>
__global__ void kTransposeSharedCoalescePadding(const int *src, int *dst, const int M, const int N)
{
    const int bx = blockIdx.x * blockSize;
    const int by = blockIdx.y * blockSize;
    const int gx = bx + threadIdx.x;
    const int gy = by + threadIdx.y;
    __shared__ int s_data[blockSize][blockSize + 1]; 
    /* 
    * 定义：相同线程束中(更严谨:相同memory transaction(load/store 128 bytes)，即可能小于32 线程)的不同线程访问相同Bank的不同地址。(只针对于共享内存)
    * 所以向量化对于Bank Conflict有影响，一个线程不同的数据load/store会有不同的Bank Conflict效果
    * 一共有32 bank，每个Bank32 bits 数据，或者64 bits
    * 避免Bank Conflict: 1.padding(改变共享内存在Bank的布局) 2.swizzling(异或特性) 3.如果可以，调整线程访问模式为广播模式 
    * https://forums.developer.nvidia.com/t/how-to-understand-the-bank-conflict-of-shared-mem/260900/2
    */

    if (gx < N && gy < M) {
        s_data[threadIdx.y][threadIdx.x] = src[gy * N + gx];    
    }
    __syncthreads();

    const int gx2 = by + threadIdx.x;
    const int gy2 = bx + threadIdx.y;

    if (gx2 < M && gy2 < N) {
        dst[gy2 * M + gx2] = s_data[threadIdx.x][threadIdx.y];
    }
}
void iTransposeSharedCoalescePadding(const int *src, int *dst, const int M, const int N, int* kernel_result) {
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(CEIL(N, BLOCK_SIZE), CEIL(M, BLOCK_SIZE));

    kTransposeSharedCoalescePadding<BLOCK_SIZE><<<gridSize, blockSize>>>(src, dst, M, N);
}


// 使用shared memory 实现内存在读写上的合并 + swizzling避免Bank Conclict
template<const int blockSize>
__global__ void kTransposeSharedCoalesceSwizzling(const int *src, int *dst, const int M, const int N)
{
    const int bx = blockIdx.x * blockSize;
    const int by = blockIdx.y * blockSize;
    const int gx = bx + threadIdx.x;
    const int gy = by + threadIdx.y;
    __shared__ int s_data[blockSize][blockSize]; 

    if (gx < N && gy < M) {
        s_data[threadIdx.y][threadIdx.x ^ threadIdx.y] = src[gy * N + gx];
    }
    __syncthreads();

    const int gx2 = by + threadIdx.x;
    const int gy2 = bx + threadIdx.y;

    if (gx2 < M && gy2 < N) {
        dst[gy2 * M + gx2] = s_data[threadIdx.x][threadIdx.x ^ threadIdx.y];
    }
}
void kTransposeSharedCoalesceSwizzling(const int *src, int *dst, const int M, const int N, int* kernel_result) {
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(CEIL(N, BLOCK_SIZE), CEIL(M, BLOCK_SIZE));

    kTransposeSharedCoalesceSwizzling<BLOCK_SIZE><<<gridSize, blockSize>>>(src, dst, M, N);
}



int main(int argc, char **argv)
{
    int repeat_times = 10;
    double iStart, iElaps;
    int N = 1 << 13; // N列
    int M = 1 << 12; // M行
    int total_size = M * N;
    float total_time;
    size_t bytes = total_size * sizeof(int);
    std::cout << "______________________With row: " << M << ", col: " << N << "______________________" << std::endl;

    // allocate host memory
    int *h_src = (int *)malloc(bytes);
    int *host_result = (int *)malloc(bytes);
    int *kernel_result = (int *)malloc(bytes);

    //  initialize host array
    for (int i = 0; i < total_size; i++)
    {
        h_src[i] = (int)(rand() & 0xFF);
    }
    // printMatrix(h_src, M, N);

    // allocate deveice memory
    int *d_src = nullptr, *d_dst = nullptr;
    CHECK(hipMalloc(reinterpret_cast<void **>(&d_src), bytes));
    CHECK(hipMalloc(reinterpret_cast<void **>(&d_dst), bytes));

    // transpose at host  
    iStart = seconds();
    transposeHost(h_src, host_result, M, N);
    iElaps = seconds() - iStart;
    std::cout << RED << "[host]: elapsed = " << iElaps * 1000 << " ms, " << RESET << std::endl << std::endl;
    // printMatrix(host_result, N, M);

    // navie
    CHECK(hipMemcpy(d_src, h_src, bytes, hipMemcpyHostToDevice));
    total_time = TIME_RECORD(repeat_times, ([&]{iTransposeNavie(d_src, d_dst, M, N, kernel_result);}));
    std::cout << RED << std::endl << "[device navie]: elapsed = " << total_time / repeat_times << " ms " << RESET << std::endl;
    hipMemcpy(kernel_result, d_dst, M * N * sizeof(int), hipMemcpyDeviceToHost);
    verifyResult(host_result, kernel_result, N, M);

    // coalesced store
    CHECK(hipMemcpy(d_src, h_src, bytes, hipMemcpyHostToDevice));
    total_time = TIME_RECORD(repeat_times, ([&]{iTransposeStoreCoalesce(d_src, d_dst, M, N, kernel_result);}));
    std::cout << RED << std::endl << "[device coalesced store]: elapsed = " << total_time / repeat_times << " ms " << RESET << std::endl;
    hipMemcpy(kernel_result, d_dst, M * N * sizeof(int), hipMemcpyDeviceToHost);
    verifyResult(host_result, kernel_result, N, M); 

    // coalesced load&store in shared memory
    CHECK(hipMemcpy(d_src, h_src, bytes, hipMemcpyHostToDevice));
    total_time = TIME_RECORD(repeat_times, ([&]{iTransposeSharedCoalesce(d_src, d_dst, M, N, kernel_result);}));
    std::cout << RED << std::endl << "[device coalesced store&load]: elapsed = " << total_time / repeat_times << " ms " << RESET << std::endl;
    hipMemcpy(kernel_result, d_dst, M * N * sizeof(int), hipMemcpyDeviceToHost);
    verifyResult(host_result, kernel_result, N, M); 

    // coalesced load&store in shared memory + padding
    CHECK(hipMemcpy(d_src, h_src, bytes, hipMemcpyHostToDevice));
    total_time = TIME_RECORD(repeat_times, ([&]{iTransposeSharedCoalescePadding(d_src, d_dst, M, N, kernel_result);}));
    std::cout << RED << std::endl << "[device coalesced store&load padding]: elapsed = " << total_time / repeat_times << " ms " << RESET << std::endl;
    hipMemcpy(kernel_result, d_dst, M * N * sizeof(int), hipMemcpyDeviceToHost);
    verifyResult(host_result, kernel_result, N, M); 

    // coalesced load&store in shared memory + swizzling
    CHECK(hipMemcpy(d_src, h_src, bytes, hipMemcpyHostToDevice));
    total_time = TIME_RECORD(repeat_times, ([&]{kTransposeSharedCoalesceSwizzling(d_src, d_dst, M, N, kernel_result);}));
    std::cout << RED << std::endl << "[device coalesced store&load swizzling]: elapsed = " << total_time / repeat_times << " ms " << RESET << std::endl;
    hipMemcpy(kernel_result, d_dst, M * N * sizeof(int), hipMemcpyDeviceToHost);
    verifyResult(host_result, kernel_result, N, M); 

    // free host and device memory
    CHECK(hipFree(d_src));
    CHECK(hipFree(d_dst));
    free(h_src);
    free(host_result);
    free(kernel_result);

    return 1;
}